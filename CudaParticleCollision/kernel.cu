#include "hip/hip_runtime.h"
﻿#include <vector>
#include <raylib.h>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <string>
#include <iostream>
#include <hip/hip_runtime_api.h>

#define SOFTENING 1e-9f

struct Particle {
    float x, y;
    float speedX = 300, speedY = 300;
    float radius = 5;

    Color color = PURPLE;

    void Draw() {
        DrawCircle(x, y, radius, color);
    }
};

__device__ bool CheckCollision(const Particle& p1, const Particle& p2) {
    float dx = p2.x - p1.x;
    float dy = p2.y - p1.y;
    float distanceSquared = dx * dx + dy * dy;
    return distanceSquared <= (p1.radius + p2.radius) * (p1.radius + p2.radius);
}

__device__ void HandleCollision(Particle& p1, Particle& p2) {
    float dx = p2.x - p1.x;
    float dy = p2.y - p1.y;
    float distance = sqrt(dx * dx + dy * dy);
    float nx = dx / distance;
    float ny = dy / distance;
    float tx = -ny;
    float ty = nx;
    float dvx = p2.speedX - p1.speedX;
    float dvy = p2.speedY - p1.speedY;
    float dpNormal = dvx * nx + dvy * ny;
    float imp = (2.0f * dpNormal) / (1 + 1);
    p1.speedX += imp * nx;
    p1.speedY += imp * ny;
    p2.speedY -= imp * ny;
    p2.speedX -= imp * nx;
    float overlap = (p1.radius + p2.radius) - distance;
    p1.x -= overlap * nx * 0.5f;
    p1.y -= overlap * ny * 0.5f;
    p2.x += overlap * nx * 0.5f;
    p2.y += overlap * ny * 0.5f;
    p1.color = RED;
    p2.color = BLUE;
}

__global__ void ParticlePhysics(Particle* particles, float dt, int n, int screenWidth, int screenHeight) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        particles[index].x += particles[index].speedX * dt;
        particles[index].y += particles[index].speedY * dt;

        // Border collision detection
        if (particles[index].y - particles[index].radius < 0) {
            particles[index].y = particles[index].radius;
            particles[index].speedY *= -1;
        }
        if (particles[index].y + particles[index].radius > screenHeight) {
            particles[index].y = screenHeight - particles[index].radius;
            particles[index].speedY *= -1;
        }
        if (particles[index].x - particles[index].radius < screenWidth / 5) {
            particles[index].x = screenWidth / 5 + particles[index].radius;
            particles[index].speedX *= -1;
        }
        if (particles[index].x + particles[index].radius > screenWidth) {
            particles[index].x = screenWidth - particles[index].radius;
            particles[index].speedX *= -1;
        }
    }
}
__global__ void ParticleCollision(Particle* particles, float dt, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {

        for (int j = 0; j < n; j++) {
            if (j != index && CheckCollision(particles[index], particles[j])) {
                HandleCollision(particles[index], particles[j]);
            }
        }
    }
}
void AddNewParticle(std::vector<Particle>& particles, Particle*& d_particles) {
    if (IsKeyPressed(KEY_SPACE)) {
        Particle newParticle;
        newParticle.x = GetScreenWidth() / 2.0f;
        newParticle.y = GetScreenHeight() / 2.0f;
        newParticle.speedX = 300;
        newParticle.speedY = 300;
        particles.push_back(newParticle);
        // Resize d_particles and copy data
        hipFree(d_particles);
        hipMallocManaged(&d_particles, particles.size() * sizeof(Particle));
        hipMemcpy(d_particles, particles.data(), particles.size() * sizeof(Particle), hipMemcpyHostToDevice);
    }
}

void CenterAllParticles(std::vector<Particle>& particles, Particle*& d_particles) {
    if (IsKeyDown(KEY_E)) {
        float centerX = (GetScreenWidth() / 5 + GetScreenWidth()) / 2.0f;
        float centerY = GetScreenHeight() / 2.0f;
        for (int i = 0; i < particles.size(); i++) {
            float dx = centerX - particles[i].x;
            float dy = centerY - particles[i].y;
            float length = sqrt(dx * dx + dy * dy);
            if (length != 0) {
                dx /= length;
                dy /= length;
            }
            particles[i].x += dx * 0.3f;
            particles[i].y += dy * 0.3f;
        }
        // Copy updated particle positions back to d_particles
        hipMemcpy(d_particles, particles.data(), particles.size() * sizeof(Particle), hipMemcpyHostToDevice);
    }
}
void Spawn50Particles(std::vector<Particle>& particles, Particle*& d_particles) {
    if (IsKeyPressed(KEY_R)) {
        for (int i = 0; i < 50; i++) {
            int RandomXPos = GetRandomValue(GetScreenWidth() / 2, GetScreenWidth());
            int RandomYPos = GetRandomValue(0, GetScreenHeight());
            Particle newParticle;
            newParticle.x = RandomXPos;
            newParticle.y = RandomYPos;
            newParticle.speedX = 300;
            newParticle.speedY = 300;
            particles.push_back(newParticle);
        }
        // Resize d_particles and copy data
        hipFree(d_particles);
        hipMallocManaged(&d_particles, particles.size() * sizeof(Particle));
        hipMemcpy(d_particles, particles.data(), particles.size() * sizeof(Particle), hipMemcpyHostToDevice);
    }
}

int main() {
    InitWindow(800, 600, "Particle Collision");
    SetTargetFPS(0);
    std::string add = "Add Particles \n\nPress: Space";
    std::string center = "\n\nCenter Particles \n\nPress: E";
    std::string spawn = "\nSpawn 50 Particles \n\nPress: R";
    std::string pDelete = "Delete Particles\n\nPress: D";

    int deviceId;
    hipGetDevice(&deviceId);

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, deviceId);

    int SMs = props.multiProcessorCount;

    std::vector<Particle> particles;
    particles.resize(1);

    Particle* d_particles;
    hipMallocManaged(&d_particles, particles.size() * sizeof(Particle));

    while (!WindowShouldClose()) {
        size_t threads = 256;
        size_t blocks = SMs * 36;

        float dt = GetFrameTime(); // Adjust as needed
        hipMemcpy(d_particles, particles.data(), particles.size() * sizeof(Particle), hipMemcpyHostToDevice);
        AddNewParticle(particles, d_particles);
        CenterAllParticles(particles, d_particles);
        Spawn50Particles(particles, d_particles);

        ParticlePhysics << <blocks, threads >> > (d_particles, dt, particles.size(), GetScreenWidth(), GetScreenHeight());
        ParticleCollision << <blocks, threads >> > (d_particles, dt, particles.size());

        hipDeviceSynchronize();

        if (IsKeyPressed(KEY_D)) {
            particles.clear();
        }


        hipMemcpy(particles.data(), d_particles, particles.size() * sizeof(Particle), hipMemcpyDeviceToHost);

        std::string counter = "Particles: " + std::to_string(particles.size());

        BeginDrawing();
        ClearBackground(WHITE);

        for (int i = 0; i < particles.size(); i++) {
            particles[i].Draw();
        }

        DrawRectangle(0, 0, GetScreenWidth() / 5, GetScreenHeight(), BLACK); //Menu Background
        DrawText(add.c_str(), GetScreenWidth() / 45, GetScreenHeight() / 15, GetScreenHeight() / 40, PURPLE);   //add
        DrawText(center.c_str(), GetScreenWidth() / 45, GetScreenHeight() / 9, GetScreenHeight() / 40, LIME);   //center
        DrawText(spawn.c_str(), GetScreenWidth() / 45, GetScreenHeight() / 4, GetScreenHeight() / 40, ORANGE);	//Spawn
        DrawText(pDelete.c_str(), GetScreenWidth() / 45, GetScreenHeight() / 2.5, GetScreenHeight() / 40, SKYBLUE);	//DELETE PARTICLES
        DrawText(counter.c_str(), GetScreenWidth() / 45, GetScreenHeight() / 2, GetScreenHeight() / 40, GOLD);	//Counter


        DrawFPS(10, 10);
        EndDrawing();
    }

    hipFree(d_particles);
    CloseWindow();
    return 0;
}